
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

int main() {
    hipblasHandle_t handle;
    float n = 2;
    float *A, *B, *C;
    float *dev_A, *dev_B, *dev_C;
    // Initialising the cuBLAS environment
    hipblasCreate(&handle);
    // Allocating Locked Page Host Memory
    hipMalloc((void **)&A, 2 * 2 * sizeof(float));
    hipMalloc((void **)&B, 3 * 3 * sizeof(float));
    hipMalloc((void **)&C, 3 * 3 * sizeof(float));
    for (int i = 0; i < 2 * 2; i++) {
        A[i] = 1.0f;
    }
    for (int i = 0; i < 3 * 3; i++) {
        B[i] = 1.0f;
    }   
    hipMalloc((void **)&dev_A, n * n * sizeof(float));
    hipMalloc((void **)&dev_B, n * n * sizeof(float));
    hipMalloc((void **)&dev_C, n * n * sizeof(float));
    // Copying data to device
    hipMemcpy(dev_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, n * n * sizeof(float), hipMemcpyHostToDevice);
    float alpha = 1.0f;
    float beta = 0.0f;
    // C = A * B
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, dev_A, 2, dev_B, 3, &beta, dev_C, 3);
    // Copy results back to host
    hipMemcpy(C, dev_C, 4 * 4 * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 4 * 4; i++) {
        assert(C[i] == 4.0f);  
    }
    // Liquidation of resources
    hipblasDestroy(handle);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}
