
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <assert.h>

__global__ void simpleKernel(int *input, int *output, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        output[idx] = input[idx] + 10.0;
    }
    
    if (idx == 0) {
        output[n] = input[n]; 
    }
}

int main() {
    const int n = 64;
    int *dev_input, *dev_output;
    int input[n], output[n+1]; 

    hipMalloc((void**)&dev_input, n * sizeof(int));
    hipMalloc((void**)&dev_output, (n + 1) * sizeof(int)); 

    for (int i = 0; i < n; i++) {
        input[i] = int(i);
    }

    hipMemcpy(dev_input, input, n * sizeof(int), hipMemcpyHostToDevice);
    //simpleKernel<<<1, 64>>>(dev_input, dev_output, n);
    hipDeviceSynchronize();

    hipMemcpy(output, dev_output, (n + 1) * sizeof(int), hipMemcpyDeviceToHost);

    return 0;
}
