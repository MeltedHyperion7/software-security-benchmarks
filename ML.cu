#include <hipDNN.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <assert.h>
//#include <generated_cuda_runtime_api_meta.h>

int main()
{
    hipdnnHandle_t handle;
    hipdnnCreate(&handle);

    hipdnnTensorDescriptor_t xDesc;
    hipdnnCreateTensorDescriptor(&xDesc);
    hipdnnSetTensor4dDescriptor(
      xDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 3, 3);

    hipdnnFilterDescriptor_t wDesc;
    hipdnnCreateFilterDescriptor(&wDesc);
    hipdnnSetFilter4dDescriptor(
      wDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, 1, 3, 4);

    hipdnnConvolutionDescriptor_t convDesc;
    hipdnnCreateConvolutionDescriptor(&convDesc);
    hipdnnSetConvolution2dDescriptor(
      convDesc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    hipdnnTensorDescriptor_t yDesc;
    hipdnnCreateTensorDescriptor(&yDesc);
    int n, c, h, w;
    hipdnnGetConvolution2dForwardOutputDim(convDesc, xDesc, wDesc, &n, &c, &h, &w);
    hipdnnSetTensor4dDescriptor(
    yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w);

    float alpha = 1.0f, beta = 0.0f;
    float* img;
    hipMalloc((void**)&img, sizeof(float) * 9);  // Allocate memory for the image
    hipMemcpy(img, (float[]){1, 2, 3, 4, 5, 6, 7, 8, 9}, sizeof(float) * 9, hipMemcpyHostToDevice);

    float* kernel;
    hipMalloc((void**)&kernel, sizeof(float) * 4);  // Allocate memory for the kernel
    hipMemcpy(kernel, (float[]){1, 2, 3, 4}, sizeof(float) * 4, hipMemcpyHostToDevice);

    float* out;
    hipMalloc((void**)&out, sizeof(float) * 4);  // Allocate memory for the output


    float* leak;
    hipMalloc((void**)&leak, sizeof(float) * 100);  

 hipdnnConvolutionForward(
    handle,
    &alpha,
    xDesc,
    img,
    wDesc,
    kernel,
    convDesc,
    HIPDNN_CONVOLUTION_FWD_ALGO_GEMM,
    0,
    0,
    &beta,
    yDesc,
    out,
    1,
    3,
    2,
    1,
    0);

    // Cleanup (intentionally not freeing 'leak')


    hipFree(img);
    hipFree(kernel);
    hipFree(out);

    return 0;
}
