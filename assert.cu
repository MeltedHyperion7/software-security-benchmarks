
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <assert.h>

__global__ void simpleKernel(float *input, float *output, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        output[idx] = input[idx] + 10.0f;
    }
    
    if (idx == 0) {
        output[n] = input[n]; 
    }
}

int main() {
    const int n = 64;
    float *dev_input, *dev_output;
    float input[n], output[n+1]; 

    hipMalloc((void**)&dev_input, n * sizeof(float));
    hipMalloc((void**)&dev_output, (n + 1) * sizeof(float)); 

    for (int i = 0; i < n; i++) {
        input[i] = float(i);
    }

    hipMemcpy(dev_input, input, n * sizeof(float), hipMemcpyHostToDevice);
    //simpleKernel<<<1, 64>>>(dev_input, dev_output, n);
    hipDeviceSynchronize();

    hipMemcpy(output, dev_output, (n + 1) * sizeof(float), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < n; i++) {
        assert(output[i] == input[i] + 10.0f);
    }

    

    hipFree(dev_input);
    hipFree(dev_output);
    return 0;
}
